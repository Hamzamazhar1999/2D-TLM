#include "hip/hip_runtime.h"
//Including all the libraries essentials
#include "hip/hip_runtime.h"   
#include ""
#include <iostream>
#include <fstream>  //for using output streaming to an external file
#include <iomanip>  // for setprecision
#include <time.h>   // for clock

#define M_PI 3.14276   //definition of pi value
#define c 299792458     //speed of light
#define mu0 M_PI*4e-7   //magnetic permeability
#define eta0 c*mu0      //impedance of the wave
#define square_root 1.41421356237 //square root of 2, constant defined to remove calculation again and again

#define NX 100          //first dimension of nodes (array X dimension)
#define NY 100          //second dimension of nodes (array Y dimension)
#define NT 8192          //number of time steps

/*Kernels for applying source, scattering, connecting and using probe to find the output*/
__global__ void tlmSource(double* dev_V1, double* dev_V2, double* dev_V3, double* dev_V4, int* dev_Ein, double E0);
__global__ void tlmScatter(double* dev_V1, double* dev_V2, double* dev_V3, double* dev_V4, double Z, int* dev_Ein, double E0);
__global__ void tlmConnect(double* V1, double* V2, double* V3, double* V4, double rXmin, double rXmax, double rYmin, double rYmax, int* dev_Eout);
__global__ void tlmOutput(double* dev_vout, double * dev_V2, double *dev_V4, int n, int* dev_Eout);
/*Function for declaring a 2d array using dynamic allocation*/
double** declare_2Darray(void);

using namespace std;

int main()
{
    clock_t start, end;     // defining clock variables for execution time calculation
    double cpu_time;

    // 2D mesh variables
    double I = 0, tempV = 0, E0 = 0, V = 0;
    double Z = eta0 / square_root;
    double dl = 1;      //seperation of nodes
    double dt = dl / (square_root * c);    //time step duration
    double width = 20 * dt * square_root;
    double delay = 100 * dt * square_root;

    /* Voltage Arrays for the host*/
    double** V1 = declare_2Darray();
    double** V2 = declare_2Darray();
    double** V3 = declare_2Darray();
    double** V4 = declare_2Darray();
    /* Voltage Arrays for the device*/
    double* dev_V1;
    double* dev_V2;
    double* dev_V3;
    double* dev_V4;
    /* Voltage Array dynamic memory allocation in GPU*/
    hipMalloc((void**)&dev_V1, NX * NY * sizeof(double));
    hipMalloc((void**)&dev_V2, NX * NY * sizeof(double));
    hipMalloc((void**)&dev_V3, NX * NY * sizeof(double));
    hipMalloc((void**)&dev_V4, NX * NY * sizeof(double));
    /* Voltage Arrays copied from host to device*/
    hipMemcpy(dev_V1, V1[0], NX * NY * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dev_V2, V2[0], NX * NY * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dev_V3, V3[0], NX * NY * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dev_V4, V4[0], NX * NY * sizeof(double), hipMemcpyHostToDevice);

    // boundary coefficients
    double rXmin = -1;
    double rXmax = -1;
    double rYmin = -1;
    double rYmax = -1;

    //Application of excitation voltage at this node
    int* Ein = (int*)malloc(2 * sizeof(int));
    Ein[0] = 10;
    Ein[1] = 10;
    //Monitoring Node is defined here
    int* Eout = (int*)malloc(2 * sizeof(int));
    Eout[0] = 15;
    Eout[1] = 15;
    /*defining the GPU data arrays to transfer these nodes to the GPU*/
    int* dev_Ein;
    int* dev_Eout;
    /* Simulation nodes dynamic memory allocation in GPU*/
    hipMalloc((void**)&dev_Ein, 2 * sizeof(int));
    hipMalloc((void**)&dev_Eout, 2 * sizeof(int));
    /* Simulation nodes copied from host to device*/
    hipMemcpy(dev_Ein, Ein, 2 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_Eout, Eout, 2 * sizeof(int), hipMemcpyHostToDevice);

    /*Output Voltage Array of size NT*/
    double v_output[NT];
    for (int i = 0; i < NT; i++)
    {
        v_output[i] = 0;
    }
    /*Output Array defined on GPU*/
    double* dev_vout;
    /*Output Array GPU memory allocation to the size NT*/
    hipMalloc((void**)&dev_vout, NT * sizeof(double));
    //Determining Kernel Size
    dim3 dimBlock(10, 10);
    dim3 dimGrid(ceil(NX / dimBlock.x), ceil(NY / dimBlock.y));
    //Creating an output file to log output voltages against the n*dt
    ofstream output("output_usingGPU.out");
    //Starting the TIME CALCULATION here for the actual loop for calculation
    start = clock();
    for (int n = 0; n < NT; n++) // Loop runs for the total time steps defined above
    {
        E0 = (1 / square_root) * exp(-(n * dt - delay) * (n * dt - delay) / (width * width));//Excitation Voltage 
        //Source Application Kernel
        tlmSource<<<1,1>>>(dev_V1, dev_V2,dev_V3, dev_V4, dev_Ein, E0);
        // Scattering Kernel
        tlmScatter << <dimGrid, dimBlock >> > (dev_V1, dev_V2, dev_V3, dev_V4, Z, dev_Ein,E0);
        // Connecting Kernel 
        tlmConnect << <dimGrid, dimBlock >> > (dev_V1, dev_V2, dev_V3, dev_V4, rXmin, rXmax, rYmin, rYmax,dev_Eout);
        // Output Probing Kernel 
        tlmOutput << <1, 1 >> > (dev_vout, dev_V2, dev_V4, n, dev_Eout);
    }
    /*Output Array from the Probing kernel is copied back to device for logging after loop ends*/
    hipMemcpy(v_output, dev_vout, NT *  sizeof(double), hipMemcpyDeviceToHost);
    //Printing outputs to an output file
    for (int n = 0; n < NT; n++) 
    {
        output << " n*dt" << "\t\t" << "Output" << endl;
        output << n * dt << "\t\t" << v_output[n] << endl;
    }
    end = clock();//ending the TIME CALCULATION here
    output.close();
    /*Freeing the defined variables and arrays*/
    hipFree(dev_V1);
    hipFree(dev_V2);
    hipFree(dev_V3);
    hipFree(dev_V4);
    hipFree(dev_Ein);
    hipFree(dev_Eout);
    hipFree(dev_vout);

    /*Printing Time calculation for the main 2d algorithm here*/
    double TLM_Execution_Time = double(end - start) / double(CLOCKS_PER_SEC);
    cout << "Time taken by TLM algorithm using GPU is : " << fixed << TLM_Execution_Time << setprecision(5);
    cout << " sec " << endl;
    return 0;
}
//definition of the Source Kernel 
__global__ void tlmSource(double* dev_V1, double* dev_V2, double* dev_V3, double* dev_V4, int* dev_Ein, double E0)
{
    //Source Application is done with pointer-to-pointer ensuring indexing
    dev_V1[dev_Ein[0] * NX + dev_Ein[1]] = dev_V1[dev_Ein[0] * NX + dev_Ein[1]] + E0;
    dev_V2[dev_Ein[0] * NX + dev_Ein[1]] = dev_V2[dev_Ein[0] * NX + dev_Ein[1]] - E0;
    dev_V3[dev_Ein[0] * NX + dev_Ein[1]] = dev_V3[dev_Ein[0] * NX + dev_Ein[1]] - E0;
    dev_V4[dev_Ein[0] * NX + dev_Ein[1]] = dev_V4[dev_Ein[0] * NX + dev_Ein[1]] + E0;
}
//Definition of the scatter kernel
__global__ void tlmScatter(double* dev_V1, double* dev_V2, double* dev_V3, double* dev_V4, double Z,int* dev_Ein,double E0 )
{
    /*running on threads and blocks for most optimal application*/
    int x = blockIdx.x * blockDim.x + threadIdx.x; 
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    /*Indexing over here is done to enssure pointer-to-pointer objects,
      so for an array dev_V1[x][y], the indexing is done like this: 
      dev_V1[x*NX+y].
     */
    if ((x*NX+y)<NX*NY) // breaking the CPU code into if statements for parallelization
    {
        double I = (2 * dev_V1[x * NX + y] + 2 * dev_V4[x * NX + y] - 2 * dev_V2[x * NX + y] - 2 * dev_V3[x * NX + y]) / (4 * Z);

        double V = 2 * dev_V1[x * NX + y] - I * Z;         // port1
        dev_V1[x * NX + y] = V - dev_V1[x * NX + y];
        V = 2 * dev_V2[x * NX + y] + I * Z;         // port2
        dev_V2[x * NX + y] = V - dev_V2[x * NX + y];
        V = 2 * dev_V3[x * NX + y] + I * Z;         // port3
        dev_V3[x * NX + y] = V - dev_V3[x * NX + y];
        V = 2 * dev_V4[x * NX + y] - I * Z;         // port4
        dev_V4[x * NX + y] = V - dev_V4[x * NX + y];
    }
}
//Definition of the connect kernel
__global__ void tlmConnect(double* dev_V1, double* dev_V2, double* dev_V3, double* dev_V4, double rXmin, double rXmax, double rYmin, double rYmax, int* dev_Eout)
{
    /*running on threads and blocks for most optimal application*/
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x > 0 && (x * NX + y) < NX*NY)//Again, breaking the for loops from CPU code into if statements for parallelization
    {
        double tempV = dev_V2[x * NX + y];
        dev_V2[x * NX + y] = dev_V4[(x - 1) * NX + y];
        dev_V4[(x - 1) * NX + y] = tempV;
    }

    if (y > 0 && (x * NX + y) < NX * NY)
    {
        double tempV = dev_V1[x * NX + y];
        dev_V1[x * NX + y] = dev_V3[x * NX + (y - 1)];
        dev_V3[x * NX + (y - 1)] = tempV;
    }
    //Boundary Conditions applied here
    if (x < NX && y == NX - 1) // for maximum Y node dimension value
    {
        dev_V3[x * NX + (NY-1)] = rYmax * dev_V3[x * NX + (NY - 1)];
    }

    if (x < NX && y == 0) // for minimum Y node dimension value
    {
        dev_V1[x * NX + y] = rYmin * dev_V1[x*NX ];
    }

    if (x == NX - 1 && y < NX)// for maximum X node dimension value
    {
        dev_V4[(NX-1)*NX+y] = rXmax * dev_V4[(NX - 1) * NX + y];
    }

    if (x == 0 && y < NX)// for minimum X node dimension value
    {
        dev_V2[y] = rXmin * dev_V2[y];
    }
}
//Definition of Output Probing Kernel 
__global__ void tlmOutput(double* dev_vout, double* dev_V2, double* dev_V4, int n, int* dev_Eout)
{
    //probing the output as a sum of the dev_v2 and dev_V4 at 15,15 node using the Eout values
    dev_vout[n] = dev_V2[dev_Eout[0] + dev_Eout[1] * NX] + dev_V4[dev_Eout[0] + dev_Eout[1] * NX];
}
//2D dynamic allocation of the ararys 
double** declare_2Darray()
{
    double** V = new double* [NX];
    V[0] = new double[NX * NY];
    for (int i = 1; i < NX; ++i)
    {
        V[i] = V[i - 1] + NY;
    }
    for (int i = 0; i < NX; ++i) {
        for (int j = 0; j < NY; ++j) {
            V[i][j] = 0;
        }
    }
    return V;
}
